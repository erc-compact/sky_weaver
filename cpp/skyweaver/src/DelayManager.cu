#include "psrdada_cpp/cuda_utils.hpp"
#include "skyweaver/DelayManager.cuh"

#include <cerrno>
#include <cstring>
#include <errno.h>
#include <fstream>
#include <iostream>
#include <sstream>

namespace skyweaver
{

DelayManager::DelayManager(std::string delay_file, hipStream_t stream)
    : _copy_stream(stream)
{
  BOOST_LOG_TRIVIAL(debug) << "Constructing new DelayManager instance";
  BOOST_LOG_TRIVIAL(debug) << "Opening delay model file: " << delay_file;
  _input_stream.open(delay_file, std::ios::in | std::ios::binary);
  if(!_input_stream.is_open()) {
    std::ostringstream error_msg;
    error_msg << "Error: Unable to open file delay file due to error: "
              << std::strerror(errno);
    throw std::runtime_error(error_msg.str().c_str());
  }
  BOOST_LOG_TRIVIAL(debug) << "Delay model file successfully opened";

  // Get first model from file
  read_next_model();
}

DelayManager::~DelayManager()
{
  if(_input_stream.is_open()) {
    _input_stream.close();
  }
}

DelayManager::DelayVectorDType const& DelayManager::delays(double epoch)
{
  // This function should return the delays in GPU memory

  // Scan through the model file until we reach model that
  // contains valid delays for the given epoch or until we
  // hit EOF (which throws an exception).
  if (epoch < _header.start_epoch)
  {
    throw InvalidDelayEpoch(epoch);
  }
  
  while(!validate_model(epoch)) { read_next_model(); }
  
  thrust::copy(_delays_h.begin(), _delays_h.end(), _delays_d.begin());
  return _delays_d;
}

bool DelayManager::validate_model(double epoch) const
{
  return ((epoch >= _header.start_epoch) && (epoch <= _header.end_epoch));
}

void DelayManager::safe_read(char* buffer, std::size_t nbytes)
{
  BOOST_LOG_TRIVIAL(debug) << "At byte " << _input_stream.tellg() << " of the input file";
  _input_stream.read(buffer, nbytes);
  if(_input_stream.eof()) {
    // Reached the end of the delay model file
    // TODO: Decide what the behaviour should be here.
    throw std::runtime_error("Reached end of delay model file");
  } else if (_input_stream.fail() || _input_stream.bad()) {
    std::ostringstream error_msg;
    error_msg << "Error: Unable to read from delay file: "
              << std::strerror(errno);
    throw std::runtime_error(error_msg.str().c_str());
  }
}

void DelayManager::read_next_model()
{
  BOOST_LOG_TRIVIAL(debug) << "Reading delay model from file";
  // Read the model header
  safe_read(reinterpret_cast<char*>(&_header), sizeof(_header));

  BOOST_LOG_TRIVIAL(debug) << "Delay model read successful";
  BOOST_LOG_TRIVIAL(debug) << "Delay model parameters: "
                           << "Nantennas = " << _header.nantennas << ", "
                           << "Nbeams = " << _header.nbeams << ", "
                           << "Start = " << _header.start_epoch << ", "
                           << "End = " << _header.end_epoch;

  // Resize the arrays for the delay model on the host and GPU
  const std::size_t nelements = _header.nantennas * _header.nbeams;
  _delays_h.resize(nelements);
  _delays_d.resize(nelements);

  // Read the weight, offset, rate tuples from the file
  safe_read(reinterpret_cast<char*>(thrust::raw_pointer_cast(_delays_h.data())),
            nelements * sizeof(DelayVectorHType::value_type));
}

} // namespace skyweaver