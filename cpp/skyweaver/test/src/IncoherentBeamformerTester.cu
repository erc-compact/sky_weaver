#include "hip/hip_runtime.h"
#include "psrdada_cpp/common.hpp"
#include "psrdada_cpp/cuda_utils.hpp"
#include "skyweaver/beamformer_utils.cuh"
#include "skyweaver/skyweaver_constants.hpp"
#include "skyweaver/test/IncoherentBeamformerTester.cuh"

#include <cmath>
#include <complex>
#include <random>

namespace skyweaver
{
namespace test
{

IncoherentBeamformerTester::IncoherentBeamformerTester()
    : ::testing::Test(), _stream(0)
{
}

IncoherentBeamformerTester::~IncoherentBeamformerTester()
{
}

void IncoherentBeamformerTester::SetUp()
{
    CUDA_ERROR_CHECK(hipStreamCreate(&_stream));
}

void IncoherentBeamformerTester::TearDown()
{
    CUDA_ERROR_CHECK(hipStreamDestroy(_stream));
}

void IncoherentBeamformerTester::beamformer_c_reference(
    HostVoltageVectorType const& taftp_voltages,
    HostRawPowerVectorType& tf_powers_raw,
    HostPowerVectorType& tf_powers,
    int nchannels,
    int tscrunch,
    int fscrunch,
    int ntimestamps,
    int nantennas,
    int nsamples_per_timestamp,
    HostScalingVectorType const& scale,
    HostScalingVectorType const& offset)
{
    static_assert(SKYWEAVER_NPOL == 2, "Tests only work for dual poln data.");
    const int tp        = nsamples_per_timestamp;
    const int ftp       = nchannels * tp;
    const int aftp      = nantennas * ftp;
    double power_sum    = 0.0;
    double power_sq_sum = 0.0;
    std::size_t count   = 0;

    char4 const* taftp_voltages_c4 =
        (char4 const*)thrust::raw_pointer_cast(taftp_voltages.data());
    for(int timestamp_idx = 0; timestamp_idx < ntimestamps; ++timestamp_idx) {
        for(int subint_idx = 0; subint_idx < nsamples_per_timestamp / tscrunch;
            ++subint_idx) {
            int subint_start = subint_idx * tscrunch;
            for(int subband_idx = 0; subband_idx < nchannels / fscrunch;
                ++subband_idx) {
                int subband_start = subband_idx * fscrunch;
                {
                    float power = 0.0f;
                    for(int antenna_idx = 0; antenna_idx < nantennas;
                        ++antenna_idx) {
                        for(int channel_idx = subband_start;
                            channel_idx < subband_start + fscrunch;
                            ++channel_idx) {
                            for(int sample_idx = subint_start;
                                sample_idx < subint_start + tscrunch;
                                ++sample_idx) {
                                int input_index = timestamp_idx * aftp +
                                                  antenna_idx * ftp +
                                                  channel_idx * tp + sample_idx;
                                char4 ant = taftp_voltages_c4[input_index];
                                hipFloatComplex p0 =
                                    make_hipFloatComplex((float)ant.x,
                                                        (float)ant.y);
                                hipFloatComplex p1 =
                                    make_hipFloatComplex((float)ant.z,
                                                        (float)ant.w);
                                power += calculate_stokes(p0, p1);
                            }
                        }
                    }
                    int time_idx =
                        timestamp_idx * nsamples_per_timestamp / tscrunch +
                        subint_idx;
                    int output_idx =
                        time_idx * nchannels / fscrunch + subband_idx;
                    power_sum += power;
                    power_sq_sum += power * power;
                    ++count;
                    tf_powers_raw[output_idx] = power;
                    float scaled_power =
                        ((power - offset[subband_idx]) / scale[subband_idx]);
                    tf_powers[output_idx] =
                        (int8_t)fmaxf(-127.0f, fminf(127.0f, scaled_power));
                }
            }
        }
    }
    double power_mean = power_sum / count;
    BOOST_LOG_TRIVIAL(debug) << "Average power level: " << power_mean;
    BOOST_LOG_TRIVIAL(debug)
        << "Power variance: " << power_sq_sum / count - power_mean * power_mean;
}

void IncoherentBeamformerTester::compare_against_host(
    DeviceVoltageVectorType const& taftp_voltages_gpu,
    DeviceRawPowerVectorType& tf_powers_raw_gpu,
    DevicePowerVectorType& tf_powers_gpu,
    DeviceScalingVectorType const& scaling_vector,
    DeviceScalingVectorType const& offset_vector,
    int ntimestamps)
{
    HostVoltageVectorType taftp_voltages_host = taftp_voltages_gpu;
    HostPowerVectorType tf_powers_cuda        = tf_powers_gpu;
    HostRawPowerVectorType tf_powers_raw_cuda = tf_powers_raw_gpu;
    HostScalingVectorType h_scaling_vector    = scaling_vector;
    HostScalingVectorType h_offset_vector     = offset_vector;
    HostRawPowerVectorType tf_powers_raw_host(tf_powers_raw_gpu.size());
    HostPowerVectorType tf_powers_host(tf_powers_gpu.size());
    beamformer_c_reference(taftp_voltages_host,
                           tf_powers_raw_host,
                           tf_powers_host,
                           _config.nchans(),
                           _config.ib_tscrunch(),
                           _config.ib_fscrunch(),
                           ntimestamps,
                           _config.nantennas(),
                           _config.nsamples_per_heap(),
                           h_scaling_vector,
                           h_offset_vector);
    for(int ii = 0; ii < tf_powers_host.size(); ++ii) {
        EXPECT_TRUE(std::abs(static_cast<int>(tf_powers_host[ii]) -
                             tf_powers_cuda[ii]) <= 1);
        EXPECT_TRUE(
            std::fabs((tf_powers_raw_host[ii] - tf_powers_raw_cuda[ii]) /
                      tf_powers_raw_host[ii]) <= 1e-5);
    }
}

TEST_F(IncoherentBeamformerTester, ib_representative_noise_test)
{
    float input_level = 32.0f;
    _config.output_level(32.0f);
    std::default_random_engine generator;
    std::normal_distribution<float> normal_dist(0.0, 32.0f);
    IncoherentBeamformer incoherent_beamformer(_config);
    std::size_t ntimestamps = 32;
    std::size_t input_size =
        (ntimestamps * _config.nantennas() * _config.nchans() *
         _config.nsamples_per_heap() * _config.npol());
    HostVoltageVectorType taftp_voltages_host(input_size);
    for(int ii = 0; ii < taftp_voltages_host.size(); ++ii) {
        taftp_voltages_host[ii].x =
            static_cast<int8_t>(std::lround(normal_dist(generator)));
        taftp_voltages_host[ii].y =
            static_cast<int8_t>(std::lround(normal_dist(generator)));
    }

    float ib_scale = std::pow(input_level, 2);
    float ib_dof   = 2 * _config.ib_tscrunch() * _config.ib_fscrunch() *
                   _config.nantennas() * _config.npol();
    float ib_power_offset = ib_scale * ib_dof;
    float ib_power_scaling =
        ib_scale * std::sqrt(2 * ib_dof) / _config.output_level();
    DeviceScalingVectorType scales(_config.nchans() / _config.ib_fscrunch(),
                                   ib_power_scaling);
    DeviceScalingVectorType offset(_config.nchans() / _config.ib_fscrunch(),
                                   ib_power_offset);
    DeviceVoltageVectorType taftp_voltages_gpu = taftp_voltages_host;
    DevicePowerVectorType tf_powers_gpu;
    DeviceRawPowerVectorType tf_powers_raw_gpu;
    incoherent_beamformer.beamform(taftp_voltages_gpu,
                                   tf_powers_raw_gpu,
                                   tf_powers_gpu,
                                   scales,
                                   offset,
                                   _stream);
    compare_against_host(taftp_voltages_gpu,
                         tf_powers_raw_gpu,
                         tf_powers_gpu,
                         scales,
                         offset,
                         ntimestamps);
}

} // namespace test
} // namespace skyweaver
