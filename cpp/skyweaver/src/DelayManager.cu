#include "psrdada_cpp/cuda_utils.hpp"
#include "skyweaver/DelayManager.cuh"

#include <cerrno>
#include <cstring>
#include <errno.h>
#include <fstream>
#include <iostream>
#include <sstream>

namespace skyweaver
{

DelayManager::DelayManager(std::string delay_file, hipStream_t stream)
    : _copy_stream(stream)
{
  BOOST_LOG_TRIVIAL(debug) << "Constructing new DelayManager instance";
  BOOST_LOG_TRIVIAL(debug) << "Opening delay model file: " << delay_file;
  _input_stream.open(delay_file, std::ios::in | std::ios::binary);
  if(!_input_stream.is_open()) {
    std::ostringstream error_msg;
    error_msg << "Error: Unable to open file delay file due to error: "
              << std::strerror(errno);
    throw std::runtime_error(error_msg.str().c_str());
  }
  BOOST_LOG_TRIVIAL(debug) << "Delay model file successfully opened";

  // Get first model from file
  read_next_model();
}

DelayManager::~DelayManager()
{
  if(_input_stream.is_open()) {
    _input_stream.close();
  }
  if(_input_stream.fail()) {
    BOOST_LOG_TRIVIAL(error)
        << "Unable to close delay file due to error: " << std::strerror(errno);
  }
}

DelayManager::DelayVectorDType const& DelayManager::delays(double epoch)
{
  // This function should return the delays in GPU memory

  // Scan through the model file until we reach model that
  // contains valid delays for the given epoch or until we
  // hit EOF (which throws an exception).
  while(!validate_model(epoch)) { read_next_model(); }
  thrust::copy(_delays_h.begin(), _delays_h.end(), _delays_d.begin());
  return _delays_d;
}

bool DelayManager::validate_model(double epoch) const
{
  return ((epoch >= _header.start_epoch) && (epoch <= _header.end_epoch));
}

void DelayManager::safe_read(char* buffer, std::size_t nbytes)
{
  if(_input_stream.eof()) {
    // Reached the end of the delay model file
    // TODO: Decide what the behaviour should be here.
    throw std::runtime_error("Reached end of delay model file");
  }
  _input_stream.read(buffer, nbytes);
  if(_input_stream.fail() || _input_stream.bad()) {
    std::ostringstream error_msg;
    error_msg << "Error: Unable to read from delay file: "
              << std::strerror(errno);
    throw std::runtime_error(error_msg.str().c_str());
  }
}

void DelayManager::read_next_model()
{
  // Read the model header
  safe_read(reinterpret_cast<char*>(&_header), sizeof(_header));

  // Resize the arrays for the delay model on the host and GPU
  const std::size_t nelements = _header.nantennas * _header.nbeams;
  _delays_h.resize(nelements);
  _delays_d.resize(nelements);

  // Read the weight, offset, rate tuples from the file
  safe_read(reinterpret_cast<char*>(thrust::raw_pointer_cast(_delays_h.data())),
            nelements * sizeof(DelayVectorHType::value_type));
}

} // namespace skyweaver