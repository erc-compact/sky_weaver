#include "hip/hip_runtime.h"
#include "psrdada_cpp/common.hpp"
#include "psrdada_cpp/cuda_utils.hpp"
#include "skyweaver/skyweaver_constants.hpp"
#include "skyweaver/test/CoherentBeamformerTester.cuh"

#include <cmath>
#include <complex>
#include <random>

namespace skyweaver
{
namespace test
{

CoherentBeamformerTester::CoherentBeamformerTester()
    : ::testing::Test(), _stream(0)
{
}

CoherentBeamformerTester::~CoherentBeamformerTester()
{
}

void CoherentBeamformerTester::SetUp()
{
    CUDA_ERROR_CHECK(hipStreamCreate(&_stream));
}

void CoherentBeamformerTester::TearDown()
{
    CUDA_ERROR_CHECK(hipStreamDestroy(_stream));
}

void CoherentBeamformerTester::beamformer_c_reference(
    HostVoltageVectorType const& ftpa_voltages,
    HostWeightsVectorType const& fbpa_weights,
    HostPowerVectorType& tbtf_powers,
    int nchannels,
    int tscrunch,
    int fscrunch,
    int nsamples,
    int nbeams,
    int nantennas,
    int npol,
    float const* scales,
    float const* offsets)
{
    float xx, yy, xy, yx;
    double power_sum    = 0.0;
    double power_sq_sum = 0.0;
    double ib_power_sum = 0.0;
    std::size_t count   = 0;
    for(int channel_idx = 0; channel_idx < nchannels; channel_idx += fscrunch) {
        BOOST_LOG_TRIVIAL(debug)
            << "Beamformer C reference: "
            << static_cast<int>(100.0f * (channel_idx + 1.0f) / nchannels)
            << "% complete";
        for(int sample_idx = 0; sample_idx < nsamples; sample_idx += tscrunch) {
            for(int beam_idx = 0; beam_idx < nbeams; ++beam_idx) {
                float power    = 0.0f;
                float ib_power = 0.0f;
                for(int sub_channel_idx = channel_idx;
                    sub_channel_idx < channel_idx + fscrunch;
                    ++sub_channel_idx) {
                    for(int sample_offset = 0; sample_offset < tscrunch;
                        ++sample_offset) {
                        for(int pol_idx = 0; pol_idx < npol; ++pol_idx) {
                            float2 accumulator    = {0, 0};
                            float2 ib_accumulator = {0, 0};
                            for(int antenna_idx = 0; antenna_idx < nantennas;
                                ++antenna_idx) {
                                int ftpa_voltages_idx =
                                    nantennas * npol * nsamples *
                                        sub_channel_idx +
                                    nantennas * npol *
                                        (sample_idx + sample_offset) +
                                    nantennas * pol_idx + antenna_idx;
                                char2 datum = ftpa_voltages[ftpa_voltages_idx];

                                int fbpa_weights_idx =
                                    nantennas * nbeams * sub_channel_idx +
                                    nantennas * beam_idx + antenna_idx;
                                char2 weight = fbpa_weights[fbpa_weights_idx];

                                xx = datum.x * weight.x;
                                yy = datum.y * weight.y;
                                xy = datum.x * weight.y;
                                yx = datum.y * weight.x;
                                accumulator.x += xx - yy;
                                accumulator.y += xy + yx;
                                ib_accumulator.x += datum.x * datum.x;
                                ib_accumulator.y += datum.y * datum.y;
                            }
                            float r = accumulator.x;
                            float i = accumulator.y;
                            power += (r * r + i * i);
                            ib_power += (ib_accumulator.x + ib_accumulator.y);
                        }
                    }
                }
                int tf_size =
                    SKYWEAVER_NSAMPLES_PER_HEAP * nchannels / fscrunch;
                int btf_size          = nbeams * tf_size;
                int output_sample_idx = sample_idx / tscrunch;
                int tbtf_powers_idx =
                    (output_sample_idx / SKYWEAVER_NSAMPLES_PER_HEAP *
                         btf_size +
                     beam_idx * tf_size +
                     (output_sample_idx % SKYWEAVER_NSAMPLES_PER_HEAP) *
                         nchannels / fscrunch +
                     channel_idx / fscrunch);
                power_sum += power;
                ib_power_sum += ib_power;
                power_sq_sum += power * power;
                ++count;
#if SKYWEAVER_IB_SUBTRACTION
                float powerf32 = ((power - (127.0f * 127.0f * ib_power)) /
                                  scales[channel_idx / fscrunch]);
#else
                float powerf32 = ((power - offsets[channel_idx / fscrunch]) /
                                  scales[channel_idx / fscrunch]);
#endif // SKYWEAVER_IB_SUBTRACTION
                tbtf_powers[tbtf_powers_idx] =
                    (int8_t)fmaxf(-127.0f, fminf(127.0f, powerf32));
            }
        }
    }
    double power_mean = power_sum / count;
    BOOST_LOG_TRIVIAL(debug) << "Average power level: " << power_mean;
    BOOST_LOG_TRIVIAL(debug)
        << "Power variance: " << power_sq_sum / count - power_mean * power_mean;
}

void CoherentBeamformerTester::compare_against_host(
    DeviceVoltageVectorType const& ftpa_voltages_gpu,
    DeviceWeightsVectorType const& fbpa_weights_gpu,
    DeviceScalingVectorType const& scales_gpu,
    DeviceScalingVectorType const& offsets_gpu,
    DevicePowerVectorType& btf_powers_gpu,
    int nsamples)
{
    HostVoltageVectorType ftpa_voltages_host = ftpa_voltages_gpu;
    HostWeightsVectorType fbpa_weights_host  = fbpa_weights_gpu;
    HostPowerVectorType btf_powers_cuda      = btf_powers_gpu;
    HostPowerVectorType btf_powers_host(btf_powers_gpu.size());

    HostScalingVectorType scales  = scales_gpu;
    HostScalingVectorType offsets = offsets_gpu;

    beamformer_c_reference(ftpa_voltages_host,
                           fbpa_weights_host,
                           btf_powers_host,
                           _config.nchans(),
                           _config.cb_tscrunch(),
                           _config.cb_fscrunch(),
                           nsamples,
                           _config.nbeams(),
                           _config.nantennas(),
                           _config.npol(),
                           thrust::raw_pointer_cast(scales.data()),
                           thrust::raw_pointer_cast(offsets.data()));
    for(int ii = 0; ii < btf_powers_host.size(); ++ii) {
        EXPECT_NEAR(btf_powers_host[ii], btf_powers_cuda[ii], 1);
    }
}

TEST_F(CoherentBeamformerTester, representative_noise_test)
{
#if SKYWEAVER_IB_SUBTRACTION
    BOOST_LOG_TRIVIAL(info) << "Running with IB subtraction";
#else
    BOOST_LOG_TRIVIAL(info) << "Running without IB subtraction";
#endif

    const float input_level = 32.0f;
    const double pi         = std::acos(-1);
    _config.output_level(input_level);

    float scale =
        std::pow(127.0f * input_level *
                     std::sqrt(static_cast<float>(_config.nantennas())),
                 2);
    float dof =
        2 * _config.cb_tscrunch() * _config.cb_fscrunch() * _config.npol();
    float offset_val = (scale * dof);
    float scale_val  = (scale * std::sqrt(2 * dof) / _config.output_level());

    /*
    printf("Nantennas: %d, tscrunch: %d, fscrunch: %d, npol: %d, Output level:
    %f, Input level: %f, Scale val: %f, Offset val: %f\n",
           _config.nantennas(), _config.cb_tscrunch(), _config.cb_fscrunch(),
    _config.npol(), _config.output_level(), input_level, scale_val, offset_val);
    */

    DeviceScalingVectorType scales(_config.nchans() / _config.cb_fscrunch(),
                                   scale_val);
    DeviceScalingVectorType offsets(_config.nchans() / _config.cb_fscrunch(),
                                    offset_val);

    std::default_random_engine generator;
    std::normal_distribution<float> normal_dist(0.0, input_level);
    std::uniform_real_distribution<float> uniform_dist(0.0, 2 * pi);

    CoherentBeamformer coherent_beamformer(_config);

    std::size_t ntimestamps =
        max(1L,
            8192 / (_config.nchans() / _config.cb_fscrunch()) /
                (_config.nsamples_per_heap() / _config.cb_tscrunch()));
    ntimestamps =
        max(ntimestamps,
            SKYWEAVER_CB_NSAMPLES_PER_BLOCK / _config.nsamples_per_heap());
    printf("Using %ld timestamps\n", ntimestamps);

    std::size_t input_size =
        (ntimestamps * _config.nantennas() * _config.nchans() *
         _config.nsamples_per_heap() * _config.npol());
    int nsamples = _config.nsamples_per_heap() * ntimestamps;

    std::size_t weights_size =
        _config.nantennas() * _config.nchans() * _config.nbeams();

    HostVoltageVectorType ftpa_voltages_host(input_size);
    for(int ii = 0; ii < ftpa_voltages_host.size(); ++ii) {
        ftpa_voltages_host[ii].x =
            static_cast<int8_t>(std::lround(normal_dist(generator)));
        ftpa_voltages_host[ii].y =
            static_cast<int8_t>(std::lround(normal_dist(generator)));
    }

    HostWeightsVectorType fbpa_weights_host(weights_size);
    for(int ii = 0; ii < fbpa_weights_host.size(); ++ii) {
        // Build complex weight as C * exp(i * theta).
        std::complex<double> val =
            127.0f *
            std::exp(std::complex<float>(0.0f, uniform_dist(generator)));
        fbpa_weights_host[ii].x = static_cast<int8_t>(std::lround(val.real()));
        fbpa_weights_host[ii].y = static_cast<int8_t>(std::lround(val.imag()));
    }

    DeviceVoltageVectorType ftpa_voltages_gpu = ftpa_voltages_host;
    DeviceWeightsVectorType fbpa_weights_gpu  = fbpa_weights_host;
    DevicePowerVectorType btf_powers_gpu;

    coherent_beamformer.beamform(ftpa_voltages_gpu,
                                 fbpa_weights_gpu,
                                 scales,
                                 offsets,
                                 btf_powers_gpu,
                                 _stream);
    compare_against_host(ftpa_voltages_gpu,
                         fbpa_weights_gpu,
                         scales,
                         offsets,
                         btf_powers_gpu,
                         nsamples);
}

} // namespace test
} // namespace skyweaver
