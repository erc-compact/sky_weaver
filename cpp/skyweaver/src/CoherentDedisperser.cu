#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "skyweaver/CoherentDedisperser.cuh"

#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <vector>
namespace skyweaver
{

float CoherentDedisperser::get_dm_delay(float f1, float f2, float dm)
{
    return ((1 / pow(f1 / 1000, 2.0)) - (1 / pow(f2 / 1000, 2.0))) * dm *
           0.00415;
}
void CoherentDedisperser::createConfig(CoherentDedisperserConfig& config,
                                       std::size_t fft_length,
                                       std::size_t overlap_samps,
                                       std::size_t num_coarse_chans,
                                       std::size_t num_pols,
                                       std::size_t num_antennas,
                                       float tsamp,
                                       float low_freq,
                                       float bw,
                                       std::vector<float> dms)
{
    config.fft_length       = fft_length;
    config.overlap_samps    = overlap_samps;
    config.num_coarse_chans = num_coarse_chans;
    config.num_pols         = num_pols;
    config.num_antennas     = num_antennas;
    config.tsamp            = tsamp;

    config.low_freq       = low_freq;
    config.bw             = bw;
    config.high_freq      = low_freq + bw;
    config.coarse_chan_bw = bw / num_coarse_chans;

    /* Precompute DM constants */
    config._h_dms = dms;
    config._d_dms = config._h_dms;
    config._d_dm_prefactor.resize(dms.size());
    config._d_ism_responses.resize(dms.size());
    for(int i = 0; i < dms.size(); i++) {
        config._d_ism_responses[i].resize(num_coarse_chans * fft_length);
    }

    thrust::transform(
        config._d_dms.begin(),
        config._d_dms.end(),
        config._d_dm_prefactor.begin(),
        [=] __device__(float dm) { return -1.0f * TWO_PI * DM_CONST * dm; });

    for(int idx = 0; idx < config._d_dm_prefactor.size(); idx++) {
        BOOST_LOG_TRIVIAL(info)
            << "DM Prefactor: " << config._d_dm_prefactor[idx];
    }

    config.fine_chan_bw = config.coarse_chan_bw / config.fft_length;

    for(int idx = 0; idx < config._d_dm_prefactor.size(); idx++) {
        get_dm_responses(config,
                         config._d_dm_prefactor[idx],
                         config._d_ism_responses[idx]);
    }

    // data is FTPA order, we will loop over F, so we are left with TPA order.
    // Let's fuse PA to X, so TX order.
    //  We stride and batch over X and transform T
    std::size_t X  = config.num_pols * config.num_antennas;
    int n[1]       = {static_cast<int>(fft_length)}; // FFT size
    int inembed[1] = {static_cast<int>(fft_length)};
    int onembed[1] = {static_cast<int>(fft_length)};
    int istride    = X;
    int ostride    = X;
    int idist      = 1;
    int odist      = 1;
    int batch      = X;

    if(hipfftPlanMany(&config._fft_plan,
                     1,
                     n,
                     inembed,
                     istride,
                     idist,
                     onembed,
                     ostride,
                     odist,
                     HIPFFT_C2C,
                     batch) != HIPFFT_SUCCESS) {
        std::runtime_error("CUFFT error: Plan creation failed");
    }

    BOOST_LOG_TRIVIAL(debug) << "FFT plan created";
}

/**
 **/
namespace
{
#define NCHANS_PER_BLOCK 128
} // namespace
void CoherentDedisperser::dedisperse(
    thrust::device_vector<char2> const& d_tpa_voltages_in,
    thrust::device_vector<char2>& d_ftpa_voltages_out,
    std::size_t out_offset,
    int dm_idx)
{
    d_fpa_spectra.resize(d_tpa_voltages_in.size(), {0.0f, 0.0f});
    d_tpa_voltages_temp.resize(d_tpa_voltages_in.size(), {0.0f, 0.0f});
    d_tpa_voltages_out_temp.resize(d_tpa_voltages_in.size(), {0.0f, 0.0f});

    hipError_t error;

    thrust::transform(d_tpa_voltages_in.begin(),
                      d_tpa_voltages_in.end(),
                      d_tpa_voltages_temp.begin(),
                      [=] __device__(char2 const& val) {
                          hipfftComplex complex_val;
                          complex_val.x = val.x;
                          complex_val.y = val.y;
                          return complex_val;
                      });
    error = hipGetLastError();
    if(error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
    }
    BOOST_LOG_TRIVIAL(debug) << "Transformed voltages to hipfftComplex";

    hipfftExecC2C(config._fft_plan,
                 thrust::raw_pointer_cast(d_tpa_voltages_temp.data()),
                 thrust::raw_pointer_cast(d_fpa_spectra.data()),
                 HIPFFT_FORWARD);

    BOOST_LOG_TRIVIAL(debug) << "Executed forward FFT";

    // for(int i=0; i < 10; i++) {
    //     hipfftComplex x = d_fpa_spectra[i];
    //     BOOST_LOG_TRIVIAL(debug) << "d_fpa_spectra[" << i << "].x: " <<x.x;
    //     BOOST_LOG_TRIVIAL(debug) << "d_fpa_spectra[" << i << "].y: " << x.y;
    // }

    multiply_by_chirp(d_fpa_spectra,
                      d_fpa_spectra,
                      dm_idx); // operating in place.

    // for(int i=0; i < 10; i++) {
    //     hipfftComplex x = d_fpa_spectra[i];
    //     BOOST_LOG_TRIVIAL(debug) << "after d_fpa_spectra[" << i << "].x: "
    //     <<x.x; BOOST_LOG_TRIVIAL(debug) << "after d_fpa_spectra[" << i <<
    //     "].y: " << x.y;
    // }

    BOOST_LOG_TRIVIAL(debug) << "Multiplied by chirp";
    error = hipGetLastError();
    if(error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
    }

    hipfftExecC2C(config._fft_plan,
                 thrust::raw_pointer_cast(d_fpa_spectra.data()),
                 thrust::raw_pointer_cast(d_tpa_voltages_out_temp.data()),
                 HIPFFT_BACKWARD);

    error = hipGetLastError();
    if(error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
    }

    BOOST_LOG_TRIVIAL(debug) << "Executed inverse FFT";

    std::size_t N = config.fft_length;
    std::size_t discard_size =
        config.num_antennas * config.num_pols * config.overlap_samps / 2;
    BOOST_LOG_TRIVIAL(debug) << "copying from " << discard_size << " to "
                             << d_tpa_voltages_out_temp.size() - discard_size;
    // transform: divide by d_tpa_voltages_in.size()
    thrust::transform(d_tpa_voltages_out_temp.begin() + discard_size,
                      d_tpa_voltages_out_temp.end() - discard_size,
                      d_ftpa_voltages_out.begin() + out_offset,
                      [=] __device__(hipfftComplex const& val) {
                          char2 char2_val;
                          char2_val.x = static_cast<char>(
                              __float2int_rn(val.x / N)); // scale the data back
                          char2_val.y =
                              static_cast<char>(__float2int_rn(val.y / N));
                          return char2_val;
                      });

    BOOST_LOG_TRIVIAL(debug) << "Transformed hipfftComplex to char2";
}

void CoherentDedisperser::multiply_by_chirp(

    thrust::device_vector<hipfftComplex> const& d_fpa_spectra_in,
    thrust::device_vector<hipfftComplex>& d_fpa_spectra_out,
    int dm_idx)
{
    std::size_t total_chans = config._d_ism_responses[dm_idx].size();
    std::size_t batchSize   = d_fpa_spectra_in.size() / total_chans;

    if(total_chans % NCHANS_PER_BLOCK != 0) {
        throw std::runtime_error(
            "Total chans need to be a multiple of NCHANS_PER_BLOCK");
    }
    BOOST_LOG_TRIVIAL(debug) << "Total chans: " << total_chans;
    BOOST_LOG_TRIVIAL(debug) << "Batch size: " << batchSize;
    BOOST_LOG_TRIVIAL(debug) << "config._d_ism_responses[dm_idx] size: "
                             << config._d_ism_responses[dm_idx].size();
    BOOST_LOG_TRIVIAL(debug)
        << "d_fpa_spectra_in size: " << d_fpa_spectra_in.size();
    BOOST_LOG_TRIVIAL(debug)
        << "d_fpa_spectra_out size: " << d_fpa_spectra_out.size();

    dim3 blockSize(pipeline_config.nantennas() *
                   pipeline_config.npol());        // 128 threads per block
    dim3 gridSize(total_chans / NCHANS_PER_BLOCK); // 4096 blocks
    kernels::dedisperse<<<gridSize, blockSize>>>(
        thrust::raw_pointer_cast(config._d_ism_responses[dm_idx].data()),
        thrust::raw_pointer_cast(d_fpa_spectra_in.data()),
        thrust::raw_pointer_cast(d_fpa_spectra_out.data()),
        total_chans);
}
} // namespace skyweaver
namespace skyweaver
{
namespace kernels
{
__global__ void dedisperse(hipfftComplex const* __restrict__ _d_ism_response,
                           hipfftComplex const* in,
                           hipfftComplex* out,
                           unsigned total_chans)
{
    /**
    _d_ism_response is the size of 64 * 8192 = 524288
    d_fpa_spectra_in size is (65536+8192) * 64 * 2 = 9437184
    d_fpa_spectra_out size is (65536+8192) * 64 * 2 = 9437184
    **/
    const unsigned tp_size = SKYWEAVER_NANTENNAS * SKYWEAVER_NPOL; // 128

    volatile __shared__ hipfftComplex response[NCHANS_PER_BLOCK]; // 128

    const int start_chan_idx =
        blockIdx.x * NCHANS_PER_BLOCK; // 0->4096 * 128 = 0 -> 524288

    const int remainder = min(total_chans - start_chan_idx,
                              NCHANS_PER_BLOCK); // 524288 -> 0 or 128

    for(int idx = threadIdx.x; idx < remainder; idx += blockDim.x) {
        hipfftComplex const temp = _d_ism_response[start_chan_idx + idx];
        response[idx].x         = temp.x;
        response[idx].y         = temp.y;
    }

    __syncthreads();

    for(int ii = 0; ii < remainder; ++ii) {
        const int chan_idx =
            ((ii + start_chan_idx) % NCHANS_PER_BLOCK) * tp_size + threadIdx.x;
        // out[chan_idx]      = hipCmulf(response[ii], in[chan_idx]);
        out[chan_idx] = hipCmulf(response[ii], in[chan_idx]);
    }
}

struct DMResponse {
    int num_coarse_chans;
    float low_freq;
    float coarse_chan_bw;
    float fine_chan_bw;
    float dmPrefix;
    float phase_prefactor;

    DMResponse(int num_coarse_chans,
               float low_freq,
               float coarse_chan_bw,
               float fine_chan_bw,
               float dmPrefix)
        : num_coarse_chans(num_coarse_chans), low_freq(low_freq),
          coarse_chan_bw(coarse_chan_bw), fine_chan_bw(fine_chan_bw),
          dmPrefix(dmPrefix),
          phase_prefactor(fine_chan_bw * fine_chan_bw * dmPrefix)
    {
    }

    __device__ inline hipfftComplex operator()(int tid) const
    {
        int chan      = tid / num_coarse_chans; // Coarse channel
        int fine_chan = tid % num_coarse_chans; // fine channel

        float edgeFreq = low_freq + chan * coarse_chan_bw +
                         fine_chan * fine_chan_bw - fine_chan_bw * 0.5f;
        float phase = phase_prefactor / ((edgeFreq + fine_chan_bw) * edgeFreq *
                                         edgeFreq); // precalculate
        hipfftComplex weight;
        __sincosf(phase,
                  &weight.y,
                  &weight.x); // TO DO: test if it is not approximate
        return weight;
    }
};

} // namespace kernels

void get_dm_responses(CoherentDedisperserConfig& config,
                      float dm_prefactor,
                      thrust::device_vector<hipfftComplex>& response)
{
    BOOST_LOG_TRIVIAL(info) << "Generating DM responses";
    thrust::device_vector<int> indices(config.num_coarse_chans *
                                       config.fft_length);
    thrust::sequence(indices.begin(), indices.end());

    // Apply the DMResponse functor using thrust's transform
    thrust::transform(indices.begin(),
                      indices.end(),
                      response.begin(),
                      kernels::DMResponse(config.num_coarse_chans,
                                          config.low_freq,
                                          config.coarse_chan_bw,
                                          config.fine_chan_bw,
                                          dm_prefactor));
}

} // namespace skyweaver
