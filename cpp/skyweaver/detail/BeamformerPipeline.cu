#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "psrdada_cpp/cuda_utils.hpp"
#include "skyweaver/BeamformerPipeline.cuh"

#include <cstdlib>
#include <exception>
#include <stdexcept>
#include <string>

namespace skyweaver
{

template <typename CBHandler, typename IBHandler, typename StatsHandler>
BeamformerPipeline<CBHandler, IBHandler, StatsHandler>::BeamformerPipeline(
    PipelineConfig const& config,
    CBHandler& cb_handler,
    IBHandler& ib_handler,
    StatsHandler& stats_handler)
    : _config(config), _cb_handler(cb_handler), _ib_handler(ib_handler),
      _stats_handler(stats_handler), _unix_timestamp(0.0), _call_count(0)
{
    BOOST_LOG_TRIVIAL(debug) << "Constructing beanmformer pipeline";
    std::size_t nsamples = _config.gulp_length_samps();
    BOOST_LOG_TRIVIAL(debug) << "Expected gulp size: " << nsamples << " (samples)";
    if(nsamples % _config.nsamples_per_heap() != 0) {
        throw std::runtime_error("Gulp size is not a multiple of "
                                 "the number of samples per heap");
    }
    std::size_t expected_cb_size =
        (_config.nbeams() * nsamples / _config.cb_tscrunch() *
         _config.nchans() / _config.cb_fscrunch());
    _btf_cbs.resize(expected_cb_size, 0);
    BOOST_LOG_TRIVIAL(debug) << "Expected CB output size: " << expected_cb_size << " (elements)";
    std::size_t expected_ib_size = (nsamples / _config.ib_tscrunch() *
                                    _config.nchans() / _config.ib_fscrunch());
    _tf_ib.resize(expected_ib_size, 0);
    BOOST_LOG_TRIVIAL(debug) << "Expected IB output size: " << expected_ib_size << " (elements)";
    // Calculate the timestamp step per block
    _sample_clock_tick_per_block = 2 * _config.total_nchans() * nsamples;
    BOOST_LOG_TRIVIAL(debug)
        << "Sample clock tick per block: " << _sample_clock_tick_per_block;
    CUDA_ERROR_CHECK(hipStreamCreate(&_h2d_copy_stream));
    CUDA_ERROR_CHECK(hipStreamCreate(&_processing_stream));
    CUDA_ERROR_CHECK(hipStreamCreate(&_d2h_copy_stream));

    BOOST_LOG_TRIVIAL(debug) << "Constructing delay and weights managers";
    _delay_manager.reset(new DelayManager(_config, _h2d_copy_stream));
    _weights_manager.reset(new WeightsManager(_config, _processing_stream));
    _stats_manager.reset(new StatisticsCalculator(_config, _processing_stream));
    _transposer.reset(new Transposer(_config));
    _coherent_beamformer.reset(new CoherentBeamformer(_config));
    _coherent_dedisperser.reset(
        new CoherentDedisperser(_config, _dedispeser_config));
    _incoherent_beamformer.reset(new IncoherentBeamformer(_config));
    _dispenser.reset(new BufferedDispenser(_config, _processing_stream));
}

template <typename CBHandler, typename IBHandler, typename StatsHandler>
BeamformerPipeline<CBHandler, IBHandler, StatsHandler>::~BeamformerPipeline()
{
    CUDA_ERROR_CHECK(hipStreamDestroy(_h2d_copy_stream));
    CUDA_ERROR_CHECK(hipStreamDestroy(_processing_stream));
    CUDA_ERROR_CHECK(hipStreamDestroy(_d2h_copy_stream));
}

template <typename CBHandler, typename IBHandler, typename StatsHandler>
void BeamformerPipeline<CBHandler, IBHandler, StatsHandler>::init(
    ObservationHeader const& header)
{
    BOOST_LOG_TRIVIAL(debug) << "Initialising beamformer pipeline";
    _header = header;
    _cb_handler.init(_header);
    _ib_handler.init(_header);
    _stats_handler.init(_header);
    _taftp_from_host.resize(_config.gulp_length_samps() * header.nantennas * _config.nchans() * _config.npol(), {0, 0});
    BOOST_LOG_TRIVIAL(debug) << "Resized TAFTP input vector to " << _taftp_from_host.size() << " elements";
}

template <typename CBHandler, typename IBHandler, typename StatsHandler>
void BeamformerPipeline<CBHandler, IBHandler, StatsHandler>::process()
{
    BOOST_LOG_TRIVIAL(debug) << "Executing beamforming pipeline";

    // Need to add the unix timestmap to the delay manager here
    // to fetch valid delays for this epoch.
    BOOST_LOG_TRIVIAL(debug) << "Checking for delay updates";
    auto const& delays = _delay_manager->delays(_unix_timestamp);

    // Stays the same
    BOOST_LOG_TRIVIAL(debug)
        << "Calculating weights at unix time: " << _unix_timestamp;
    auto const& weights = _weights_manager->weights(delays,
                                                    _unix_timestamp,
                                                    _delay_manager->epoch());
    BOOST_LOG_TRIVIAL(debug)
        << "Transposing input data from TAFTP to FTPA order";
    _transposer->transpose(_taftp_from_host,
                           _ftpa_post_transpose,
                           _header.nantennas,
                           _processing_stream);

    // Stays the same
    BOOST_LOG_TRIVIAL(debug) << "Checking if channel statistics update request";
    _stats_manager->calculate_statistics(_ftpa_post_transpose);

    BOOST_LOG_TRIVIAL(debug) << "FTPA post transpose size: " << _ftpa_post_transpose.size();

    _dispenser->hoard(_ftpa_post_transpose);

    for(unsigned int dm_idx = 0; dm_idx < _config.coherent_dms().size();
        ++dm_idx) {
        for(unsigned int freq_idx = 0; freq_idx < _config.nchans();
            ++freq_idx) {
            BOOST_LOG_TRIVIAL(debug) << "{{{[[[<<< DM Idx: " << dm_idx << " F Idx: " << freq_idx << " >>>]]]}}}";
            BOOST_LOG_TRIVIAL(debug) << "Dispensing some voltages";
            auto const& tpa_voltages = _dispenser->dispense(freq_idx);
            BOOST_LOG_TRIVIAL(debug) << "Attempting to segfault";
            /*
            _coherent_dedisperser->dedisperse(tpa_voltages,
                                              _ftpa_dedispersed,
                                              freq_idx * _ftpa_post_transpose.size() / _config.nchans(),
                                              dm_idx);
            */
            _ftpa_dedispersed.resize(_ftpa_post_transpose.size());
            BOOST_LOG_TRIVIAL(debug) << "segfault failed";                                            
        }
        // TODO: Scalings need to be calculated based on the effective 
        // TODO: number of antennas
        auto const& ib_scaling = _stats_manager->ib_scaling();
        auto const& ib_offsets  = _stats_manager->ib_offsets();
        _incoherent_beamformer->beamform(_ftpa_dedispersed,
                                        _tf_ib_raw,
                                        _tf_ib,
                                        ib_scaling,
                                        ib_offsets,
                                        _processing_stream);

        // TODO: Scalings need to be calculated based on the effective 
        // TODO: number of antennas
        auto const& cb_scaling = _stats_manager->cb_scaling();
        auto const& cb_offsets  = _stats_manager->cb_offsets();
        _coherent_beamformer->beamform(_ftpa_dedispersed,
                                    weights,
                                    cb_scaling,
                                    cb_offsets,
                                    _tf_ib_raw,
                                    _btf_cbs,
                                    _processing_stream);
        _cb_handler(_btf_cbs);
        _ib_handler(_tf_ib);
        _stats_handler(_stats_manager->statistics());
    }
}

template <typename CBHandler, typename IBHandler, typename StatsHandler>
bool BeamformerPipeline<CBHandler, IBHandler, StatsHandler>::operator()(
    HostVoltageVectorType const& taftp_on_host)
{
    BOOST_LOG_TRIVIAL(debug) << "Pipeline operator() called";
    BOOST_LOG_TRIVIAL(debug) << "taftp_on_host size: " << taftp_on_host.size();

    if(taftp_on_host.size() != _taftp_from_host.size()) {
        throw std::runtime_error(
            std::string("Unexpected buffer size, expected ") +
            std::to_string(taftp_on_host.size()) + " but got " +
            std::to_string(_taftp_from_host.size()));
    }
    
    CUDA_ERROR_CHECK(hipMemcpyAsync(
        static_cast<void*>(thrust::raw_pointer_cast(_taftp_from_host.data())),
        static_cast<void const*>(thrust::raw_pointer_cast(taftp_on_host.data())),
        taftp_on_host.size() * sizeof(char2),
        hipMemcpyHostToDevice,
        _h2d_copy_stream));
    CUDA_ERROR_CHECK(hipStreamSynchronize(_h2d_copy_stream));

    // Calculate the unix timestamp for the block that is about to be processed
    _unix_timestamp =
        _header.utc_start +
        static_cast<long double>(_call_count * _sample_clock_tick_per_block) /
            _header.sample_clock;
    process();
    CUDA_ERROR_CHECK(hipStreamSynchronize(_processing_stream));
    CUDA_ERROR_CHECK(hipStreamSynchronize(_d2h_copy_stream));
    ++_call_count;
    return false;
}

} // namespace skyweaver