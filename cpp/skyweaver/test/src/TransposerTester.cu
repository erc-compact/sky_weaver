#include "psrdada_cpp/cuda_utils.hpp"
#include "skyweaver/skyweaver_constants.hpp"
#include "skyweaver/test/TransposerTester.cuh"

namespace skyweaver
{
namespace test
{

TransposerTester::TransposerTester(): ::testing::TestWithParam<std::size_t>(), _stream(0)
{
}

TransposerTester::~TransposerTester()
{
}

void TransposerTester::SetUp()
{
    CUDA_ERROR_CHECK(hipStreamCreate(&_stream));
}

void TransposerTester::TearDown()
{
    CUDA_ERROR_CHECK(hipStreamDestroy(_stream));
}

void TransposerTester::transpose_c_reference(HostVoltageType const& input,
                                             HostVoltageType& output,
                                             int input_nantennas,
                                             int output_nantennas,
                                             int nchans,
                                             int ntimestamps)
{
    // TAFTP to FTPA
    // Input dimensions
    int tp   = _config.nsamples_per_heap() * _config.npol();
    int ftp  = nchans * tp;
    int aftp = input_nantennas * ftp;

    // Output dimensions
    int pa  = _config.npol() * output_nantennas;
    int tpa = _config.nsamples_per_heap() * ntimestamps * pa;
    output.resize(nchans * tpa, {0, 0});

    for(int timestamp_idx = 0; timestamp_idx < ntimestamps; ++timestamp_idx) {
        for(int antenna_idx = 0; antenna_idx < input_nantennas; ++antenna_idx) {
            int input_antenna_idx = antenna_idx;
            for(int chan_idx = 0; chan_idx < nchans; ++chan_idx) {
                for(int samp_idx = 0; samp_idx < _config.nsamples_per_heap();
                    ++samp_idx) {
                    for(int pol_idx = 0; pol_idx < _config.npol(); ++pol_idx) {
                        int input_idx =
                            (timestamp_idx * aftp + input_antenna_idx * ftp +
                             chan_idx * tp + samp_idx * _config.npol() +
                             pol_idx);
                        int output_sample_idx =
                            timestamp_idx * _config.nsamples_per_heap() +
                            samp_idx;
                        int output_idx =
                            (chan_idx * tpa + output_sample_idx * pa +
                             pol_idx * output_nantennas + antenna_idx);
                        output[output_idx] = input[input_idx];
                    }
                }
            }
        }
    }
}

void TransposerTester::compare_against_host(DeviceVoltageType const& gpu_input,
                                            DeviceVoltageType const& gpu_output,
                                            std::size_t input_nantennas,
                                            std::size_t ntimestamps)
{
    HostVoltageType host_input = gpu_input;
    HostVoltageType host_output;
    HostVoltageType cuda_output = gpu_output;
    transpose_c_reference(host_input,
                          host_output,
                          input_nantennas,
                          _config.nantennas(),
                          _config.nchans(),
                          ntimestamps);
    for(int ii = 0; ii < host_output.size(); ++ii) {
        ASSERT_EQ(host_output[ii].x, cuda_output[ii].x);
        ASSERT_EQ(host_output[ii].y, cuda_output[ii].y);
    }
}

TEST_P(TransposerTester, cycling_prime_test)
{
    Transposer transposer(_config);
    std::size_t ntimestamps     = 12;
    std::size_t input_nantennas = GetParam();
    std::size_t input_size = (ntimestamps * input_nantennas * _config.nchans() *
                              _config.nsamples_per_heap() * _config.npol());

    HostVoltageType host_gpu_input(input_size);
    for(int ii = 0; ii < input_size; ++ii) {
        host_gpu_input[ii].x = (ii % 113);
        host_gpu_input[ii].y = (ii % 107);
    }
    DeviceVoltageType gpu_input = host_gpu_input;
    DeviceVoltageType gpu_output;
    transposer.transpose(gpu_input, gpu_output, input_nantennas, _stream);
    compare_against_host(gpu_input, gpu_output, input_nantennas, ntimestamps);
}

INSTANTIATE_TEST_SUITE_P(TransposerTesterSuite,
                         TransposerTester,
                         ::testing::Range<std::size_t>(1, SKYWEAVER_NANTENNAS, 1));

} // namespace test
} // namespace skyweaver
